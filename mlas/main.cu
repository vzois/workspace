#include "../../common/CudaHelper.h"
#include "../../common/ArgParser.h"
#include "mlas_core/mlas_matop.h"
#include "mlas_core/mlas_config.h"

int multiplier = 1;

void mm_test(){
	cutil::setActiveDevice(0);
	float *dA,*dB, *dC;
	uint64_t m = 1024 * multiplier;
	uint64_t n = 1024 * multiplier;
	uint64_t k = 1024 * multiplier;

	cutil::allocDevMem<float,uint64_t>(&dA,sizeof(float) * m*n, "Error allocating device memory for dA");
	cutil::allocDevMem<float,uint64_t>(&dB,sizeof(float) * n*k, "Error allocating device memory for dB");
	cutil::allocDevMem<float,uint64_t>(&dC,sizeof(float) * m*k, "Error allocating device memory for dC");

	cutil::cudaRandInit<float,unsigned int>(dA,m*n);
	cutil::cudaRandInit<float,unsigned int>(dB,n*k);

	dim3 mgrid((m-1)/TILE + 1, (k-1)/4 + 1, 1);
	dim3 mblock(TILE,4,1);

	cutil::print_grid(mgrid,mblock);

	Time<millis> t;
	t.start();
	mlas::sgemm<float,unsigned int,TILE><<<mgrid,mblock>>>(dA,dB,dC,m,n,k);
	cutil::handleDeviceErrors(hipDeviceSynchronize(),"Error executing sgemm");
	double tt= t.lap();
	double GBs = ((double)( m * n * 3 * 4 )) / (1000 * 1000 * 1000);
	uint64_t flop = 2 * n * m * k;
	double gflops = ((flop)/(tt/1000))/1000000000;
	std::cout << "GBs: " << GBs << std::endl;
	std::cout << "FLOP:" << flop << std::endl;
	std::cout << "GFLOPS:" << gflops << std::endl;



	hipFree(dA); hipFree(dB); hipFree(dC);
	hipDeviceReset();
}

int main(int argc,char **argv){
	ArgParser ap;
	ap.parseArgs(argc,argv);

	if(ap.count()== 0){
		ap.menu();
		return 0;
	}

	if(ap.exists("-c")){
		printf("Multiplier:%d\n",ap.getInt("-c"));
		multiplier = ap.getInt("-c");
	}
	//va_test();
	mm_test();

	return 0;
}
